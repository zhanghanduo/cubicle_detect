#include "hip/hip_runtime.h"
/*
 * YoloObjectDetector.cpp
 *
 *  Created on: June 19, 2018
 *      Author: Zhang Handuo
 *   Institute: NTU, ST Corp Lab
 */


// yolo object detector
#include "darknet_ros/YoloObjectDetector.hpp"
#include "utils/data.h"
#include <ros/package.h>
// Check for xServer
#include <X11/Xlib.h>
#include <algorithm>

#ifdef DARKNET_FILE_PATH
std::string darknetFilePath_ = DARKNET_FILE_PATH;
#else
#error Path of darknet repository is not defined in CMakeLists.txt.
#endif

using namespace message_filters;

namespace darknet_ros {

char *cfg;
char *weights;
char *data;
char **detectionNames;

YoloObjectDetector::YoloObjectDetector(ros::NodeHandle nh, ros::NodeHandle nh_p)
    : nodeHandle_(nh),
      nodeHandle_pub(nh_p),
      numClasses_(0),
      classLabels_(0),
      rosBoxes_(0),
      rosBoxCounter_(0),
      use_grey(false),
      blnFirstFrame(true),
      globalframe(0),
      isDepthNew(false),
      is_even_crop(false)
{
  ROS_INFO("[ObstacleDetector] Node started.");

  // Read Cuda Info and ROS parameters from config file.
  if (!CudaInfo() || !readParameters()) {
    ros::requestShutdown();
  }
//  mpDetection = new Detection(this, nodeHandle_);
//  nullHog.assign(36, 0.0);
  init();

  init_disparity_method(7, 86);

//  mpDepth_gen_run = new std::thread(&Detection::Run, mpDetection);

  hog_descriptor = new Util::HOGFeatureDescriptor(8, 2, 9, 180.0);
  img_name = ros::package::getPath("cubicle_detect") + "/seq_1/f000.png";
  file_name = ros::package::getPath("cubicle_detect") + "/seq_1/results/f000.txt";
  frame_num = 0;
}

YoloObjectDetector::~YoloObjectDetector()
{
  finish_disparity_method();
    {
    boost::unique_lock<boost::shared_mutex> lockNodeStatus(mutexNodeStatus_);
    isNodeRunning_ = false;
  }
  yoloThread_.join();
  free(depth3D);
  free(x3DPosition);
  free(y3DPosition);
  free(cfg);
  free(weights);
  free(detectionNames);
  free(data);
  free(roiBoxes_);
}

bool YoloObjectDetector::CudaInfo() {

  int deviceCount, device;
  int gpuDeviceCount = 0;
  struct hipDeviceProp_t properties{};
  hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
  if (cudaResultCode != hipSuccess)
    deviceCount = 0;
  /* machine with no GPUs can still report one emulation device */
  for (device = 0; device < deviceCount; ++ device){
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999)
      ++gpuDeviceCount;
  }
  std::cout << gpuDeviceCount << " GPU CUDA device(s) found" << std::endl;

  if (gpuDeviceCount > 0) {
    std::cout << "GPU load success!" << std::endl;
    return true;
  }
  else {
    std::cout << "GPU load fail!" << std::endl;
    return false;
  }
}

bool YoloObjectDetector::readParameters()
{
  // Load common parameters.
  nodeHandle_.param("image_view/enable_opencv", viewImage_, true);
  nodeHandle_.param("image_view/wait_key_delay", waitKeyDelay_, 3);
  nodeHandle_.param("image_view/enable_console_output", enableConsoleOutput_, false);
  nodeHandle_.param("image_view/eval", enableEvaluation_, false);

  // Check if Xserver is running on Linux.
  if (XOpenDisplay(nullptr)) {
    // Do nothing!
    ROS_INFO("[YoloObjectDetector] Xserver is running.");
  } else {
    ROS_INFO("[YoloObjectDetector] Xserver is not running.");
    viewImage_ = false;
  }

  // Set vector sizes.
  nodeHandle_.param("yolo_model/detection_classes/names", classLabels_,
                    std::vector<std::string>(0));
  numClasses_ = classLabels_.size();
  rosBoxes_ = std::vector<std::vector<RosBox_> >(numClasses_);
  rosBoxCounter_ = std::vector<int>(numClasses_);

  return true;
}

void YoloObjectDetector::init()
{
  ROS_INFO("[ObstacleDetector] init().");

  // Initialize deep network of darknet.
  std::string weightsPath;
  std::string configPath;
  std::string dataPath;
  std::string configModel;
  std::string weightsModel;

  // Look up table initialization
  counter = 0;

  nodeHandle_.param<int>("min_disparity", min_disparity, 12);
  nodeHandle_.param<int>("disparity_scope", disp_size, 128);
  nodeHandle_.param<bool>("use_grey", use_grey, false);
  nodeHandle_.param<int>("scale", Scale, 1);

  // Threshold of object detection.
  float thresh;
  nodeHandle_.param("yolo_model/threshold/value", thresh, (float) 0.3);

  // Path to weights file.
  nodeHandle_.param("yolo_model/weight_file/name", weightsModel,
                    std::string("yolo_bdd_c1_165300.weights"));
  nodeHandle_.param("weights_path", weightsPath, std::string("/default"));
  weightsPath += "/" + weightsModel;
  weights = new char[weightsPath.length() + 1];
  strcpy(weights, weightsPath.c_str());

  // Path to config file.
  nodeHandle_.param("yolo_model/config_file/name", configModel, std::string("yolo_bdd_c1.cfg"));
  nodeHandle_.param("config_path", configPath, std::string("/default"));
  configPath += "/" + configModel;
  cfg = new char[configPath.length() + 1];
  strcpy(cfg, configPath.c_str());

  // Path to data folder.
  dataPath = darknetFilePath_;
  dataPath += "/data";
  data = new char[dataPath.length() + 1];
  strcpy(data, dataPath.c_str());

  // Get classes.
  detectionNames = (char**) realloc((void*) detectionNames, (numClasses_ + 1) * sizeof(char*));
  for (int i = 0; i < numClasses_; i++) {
    detectionNames[i] = new char[classLabels_[i].length() + 1];
    strcpy(detectionNames[i], classLabels_[i].c_str());
  }

  // Load network.
  setupNetwork(cfg, weights, data, thresh, detectionNames, numClasses_,
                0, nullptr, 1, 0.5, 0, 0, 0, 0);
  yoloThread_ = std::thread(&YoloObjectDetector::yolo, this);

  // Initialize publisher and subscriber.
//  std::string cameraTopicName;
//  int cameraQueueSize;
  std::string objectDetectorTopicName;
  int objectDetectorQueueSize;
  bool objectDetectorLatch;
  std::string detectionImageTopicName;
  int detectionImageQueueSize;
  bool detectionImageLatch;
  std::string obstacleBoxesTopicName;
  int obstacleBoxesQueueSize;

  nodeHandle_.param("publishers/object_detector/topic", objectDetectorTopicName,
                    std::string("found_object"));
  nodeHandle_.param("publishers/object_detector/queue_size", objectDetectorQueueSize, 1);
  nodeHandle_.param("publishers/object_detector/latch", objectDetectorLatch, false);
  nodeHandle_.param("publishers/detection_image/topic", detectionImageTopicName,
                    std::string("detection_image"));
  nodeHandle_.param("publishers/detection_image/queue_size", detectionImageQueueSize, 1);
  nodeHandle_.param("publishers/detection_image/latch", detectionImageLatch, true);

  nodeHandle_.param("publishers/obstacle_boxes/topic", obstacleBoxesTopicName,
                    std::string("/obs_map"));
  nodeHandle_.param("publishers/obstacle_boxes/queue_size", obstacleBoxesQueueSize, 1);
  nodeHandle_.param("publishers/obstacle_boxes/frame_id", pub_obs_frame_id, std::string("camera_frame"));

  objectPublisher_ = nodeHandle_pub.advertise<std_msgs::Int8>(objectDetectorTopicName,
                                                           objectDetectorQueueSize,
                                                           objectDetectorLatch);

  obstaclePublisher_ = nodeHandle_pub.advertise<obstacle_msgs::MapInfo>(
          obstacleBoxesTopicName, obstacleBoxesQueueSize);

  detectionImagePublisher_ = nodeHandle_pub.advertise<sensor_msgs::Image>(detectionImageTopicName,
                                                                       detectionImageQueueSize,
                                                                       detectionImageLatch);

}

void YoloObjectDetector:: loadCameraCalibration(const sensor_msgs::CameraInfoConstPtr &left_info,
                                               const sensor_msgs::CameraInfoConstPtr &right_info) {

  ROS_INFO_STREAM("init calibration");

  // Check if a valid calibration exists
  if (left_info->K[0] == 0.0) {
    ROS_ERROR("The camera is not calibrated");
    return;
  }

  sensor_msgs::CameraInfoPtr left_info_copy = boost::make_shared<sensor_msgs::CameraInfo>(*left_info);
  sensor_msgs::CameraInfoPtr right_info_copy = boost::make_shared<sensor_msgs::CameraInfo>(*right_info);
//  left_info_copy->header.frame_id = "stereo";
//  right_info_copy->header.frame_id = "stereo";

  // Get Stereo Camera Model from Camera Info message
  image_geometry::StereoCameraModel stereoCameraModel;
  stereoCameraModel.fromCameraInfo(left_info_copy, right_info_copy);

  // Get PinHole Camera Model from the Stereo Camera Model
  const image_geometry::PinholeCameraModel &cameraLeft = stereoCameraModel.left();
  const image_geometry::PinholeCameraModel &cameraRight = stereoCameraModel.right();

//    double data[16] = { 1, 0, 0, -left_info->P[2]/Scale, 0, 1, 0, -left_info->P[6]/Scale, 0, 0, 0, left_info->P[0], 0};
//    double data[16] = { 1, 0, 0, -322.94284058, 0, 1, 0, -232.25880432, 0, 0, 0, 922.9965, 0, 0, 0.001376324, 0};
//    Q = cv::Mat(4, 4, CV_64F, data);

  // Get rectify intrinsic Matrix (is the same for both cameras because they are rectified)
  cv::Mat projectionLeft = cv::Mat(cameraLeft.projectionMatrix());
  cv::Matx33d intrinsicLeft = projectionLeft(cv::Rect(0, 0, 3, 3));
  cv::Mat projectionRight = cv::Mat(cameraRight.projectionMatrix());
  cv::Matx33d intrinsicRight = projectionRight(cv::Rect(0, 0, 3, 3));

  u0 = left_info->K[2];
  v0 = left_info->K[5];
  focal = left_info->K[0];
  Width = left_info->width;
  Height = left_info->height;

  Width /= Scale;
  Height /= Scale;

  rem_w = Width % 4;
  rem_h = Height % 4;
  ROS_WARN("remainder width: %d | remainder height: %d", rem_w, rem_h);

  Width_crp = Width - rem_w;
  Height_crp = Height - rem_h;

  if(rem_w || rem_h)
      is_even_crop = true;

  assert(intrinsicLeft == intrinsicRight);

  const cv::Matx33d &intrinsic = intrinsicLeft;

  // Save the baseline
  stereo_baseline_ = stereoCameraModel.baseline();
  ROS_INFO_STREAM("baseline: " << stereo_baseline_);
  assert(stereo_baseline_ > 0);

  int ii;
  x3DPosition = static_cast<double **>(calloc(Width, sizeof(double *)));
  for(ii = 0; ii < Width; ii++)
    x3DPosition[ii] = static_cast<double *>(calloc(disp_size + 1, sizeof(double)));

  y3DPosition = static_cast<double **>(calloc(Height, sizeof(double *)));
  for(ii = 0; ii < Height; ii++)
    y3DPosition[ii] = static_cast<double *>(calloc(disp_size + 1, sizeof(double)));

  depth3D = static_cast<double *>(calloc(disp_size + 1, sizeof(double)));

  ObstacleDetector.Initiate(left_info_copy->header.frame_id, disp_size, stereo_baseline_, u0, v0, focal, Width, Height);


//  // get the Region Of Interests (If the images are already rectified but invalid pixels appear)
//  left_roi_ = cameraLeft.rawRoi();
//  right_roi_ = cameraRight.rawRoi();
}

cv::Mat YoloObjectDetector::getDepth(cv::Mat &leftFrame, cv::Mat &rightFrame) {

    float elapsed_time_ms;
    cv::Mat disparity_SGBM(leftFrame.size(), CV_8UC1);

    disparity_SGBM = compute_disparity_method(leftFrame, rightFrame, &elapsed_time_ms);

    isDepthNew = true;
    return disparity_SGBM;
}

void YoloObjectDetector::DefineLUTs() {

  ROS_WARN("u0: %f | v0: %f | focal: %f | base: %f | width: %d | Height: %d", u0, v0, focal, stereo_baseline_, Width_crp, Height_crp);

    for (int r=0; r<Width; r++) {
        x3DPosition[r][0]=0;
        for (int c=1; c<disp_size+1; c++) {
            x3DPosition[r][c]=(r-u0)*stereo_baseline_/c;
//        std::cout<<xDirectionPosition[r][c]<<std::endl;
        }
    }

    for (int r=0; r<Height; r++) {
//    for (int r=300; r<301; r++) {
        y3DPosition[r][0]=0;
        for (int c=1; c<disp_size+1; c++) {
            y3DPosition[r][c]=(v0-r)*stereo_baseline_/c;
//      std::cout<<r<<", "<<c<<": "<<yDirectionPosition[r][c]<<"; ";//std::endl;
        }
    }

    depth3D[0] =0;
    for( int i = 1; i < disp_size+1; ++i){
        depth3D[i]=focal*stereo_baseline_/i; //Y*dx/B
//      std::cout<<"i: "<<i<<", "<<depthTable[i]<<"; \n";
    }

}

void YoloObjectDetector::cameraCallback(const sensor_msgs::ImageConstPtr &image1,
                                        const sensor_msgs::ImageConstPtr &image2){
    ROS_DEBUG("[ObstacleDetector] Stereo images received.");

    // std::cout<<"Debug starting cameraCallBack"<<std::endl;
    cv_bridge::CvImageConstPtr cam_image1, cam_image2, cv_rgb;

    try {
        cam_image1 = cv_bridge::toCvShare(image1, sensor_msgs::image_encodings::MONO8);
        cam_image2 = cv_bridge::toCvShare(image2, sensor_msgs::image_encodings::MONO8);

        if(use_grey) {
            cv_rgb = cam_image1;
        }
        else {
            cv_rgb = cv_bridge::toCvShare(image1, sensor_msgs::image_encodings::BGR8);
        }
        image_time_ = image1->header.stamp;
        imageHeader_ = image1->header;
    } catch (cv_bridge::Exception& e) {
        ROS_ERROR("cv_bridge exception: %s", e.what());
        return;
    }

    if (cam_image1) {

        // std::cout<<"Debug inside cameraCallBack scaling image height "<<frameHeight_<<std::endl;
        {
            boost::unique_lock<boost::shared_mutex> lockImageCallback(mutexImageCallback_);
            origLeft = cam_image1->image;//cv::Mat(cam_image1->image, left_roi_);
            origRight = cam_image2->image;//cv::Mat(cam_image2->image, right_roi_);
            camImageOrig = cv_rgb->image.clone();//cv::Mat(cv_rgb->image.clone(), left_roi_);
        }
        {
            boost::unique_lock<boost::shared_mutex> lockImageStatus(mutexImageStatus_);
            imageStatus_ = true;
        }

        // std::cout<<"Debug inside cameraCallBack starting image resize"<<std::endl;
        cv::Mat left_resized, right_resized, camImageResized;

        if(Scale != 1) {
            cv::resize(origLeft, left_resized, cv::Size(Width, Height));
            cv::resize(origRight, right_resized, cv::Size(Width, Height));
            cv::resize(camImageOrig, camImageResized, cv::Size(Width, Height));
        }else{
            left_resized = origLeft.clone();
            right_resized = origRight.clone();
            camImageResized = camImageOrig;
        }

        // std::cout<<"Debug inside cameraCallBack starting image padding"<<std::endl;

//        cv::Mat left_widthAdj, right_widthAdj, camImageWidthAdj;

        if (is_even_crop) {
//            copyMakeBorder( left_resized, left_widthAdj, 0, 0, 0, rem_w, cv::BORDER_CONSTANT, 0 );
//            copyMakeBorder( right_resized, right_widthAdj, 0, 0, 0, rem_w, cv::BORDER_CONSTANT, 0 );
//            copyMakeBorder( camImageResized, camImageWidthAdj, 0, 0, 0, rem_w, cv::BORDER_CONSTANT, cv::Scalar(0,0,0) );
            left_rectified = left_resized(cv::Rect(0, 0, Width_crp, Height_crp)).clone();
            right_rectified = right_resized(cv::Rect(0, 0, Width_crp, Height_crp)).clone();
            camImageCopy_ = camImageResized(cv::Rect(0, 0, Width_crp, Height_crp)).clone();
        } else {
            left_rectified = left_resized.clone();
            right_rectified = right_resized.clone();
            camImageCopy_ = camImageResized.clone();
        }

        // cv::Mat left_heightAdj, right_heightAdj, camImageHeightAdj;
//
//        if (is_even_crop_h) {
//            copyMakeBorder( left_widthAdj, left_rectified, 0, rem_h, 0, 0, cv::BORDER_CONSTANT, 0 );
//            copyMakeBorder( right_widthAdj, right_rectified, 0, rem_h, 0, 0, cv::BORDER_CONSTANT, 0 );
//            copyMakeBorder( camImageWidthAdj, camImageCopy_, 0, rem_h, 0, 0, cv::BORDER_CONSTANT, cv::Scalar(0,0,0) );
//        } else {
//            left_rectified = left_widthAdj;
//            right_rectified = right_widthAdj;
//            camImageCopy_ = camImageWidthAdj;
//        }

//        ROS_WARN("width: %d | height: %d", left_rectified.cols, left_rectified.rows);
    }
}

bool YoloObjectDetector::publishDetectionImage(const cv::Mat& detectionImage)
{
  if (detectionImagePublisher_.getNumSubscribers() < 1)
    return false;
  cv_bridge::CvImage cvImage;
//  cvImage.header.stamp = ros::Time::now();
  cvImage.header.stamp = image_time_;
  cvImage.header.frame_id = "detection_image";
  cvImage.encoding = sensor_msgs::image_encodings::BGR8;
  cvImage.image = detectionImage;
  detectionImagePublisher_.publish(*cvImage.toImageMsg());
  ROS_DEBUG("Detection image has been published.");
  return true;
}

// double YoloObjectDetector::getWallTime()
// {
//   struct timeval time;
//   if (gettimeofday(&time, NULL)) {
//     return 0;
//   }
//   return (double) time.tv_sec + (double) time.tv_usec * .000001;
// }

int YoloObjectDetector::sizeNetwork(network *net)
{
  int i;
  int count = 0;
  for(i = 0; i < net->n; ++i){
    layer l = net->layers[i];
    if(l.type == YOLO || l.type == REGION || l.type == DETECTION){
      count += l.outputs;
    }
  }
  return count;
}

void YoloObjectDetector::rememberNetwork(network *net)
{
  int i;
  int count = 0;
  for(i = 0; i < net->n; ++i){
    layer l = net->layers[i];
    if(l.type == YOLO || l.type == REGION || l.type == DETECTION){
      memcpy(predictions_[demoIndex_] + count, net->layers[i].output, sizeof(float) * l.outputs);
      count += l.outputs;
    }
  }
}

detection *YoloObjectDetector::avgPredictions(network *net, int *nboxes)
{
  int i, j;
  int count = 0;
  fill_cpu(demoTotal_, 0, avg_, 1);
  for(j = 0; j < demoFrame_; ++j){
    axpy_cpu(demoTotal_, 1./demoFrame_, predictions_[j], 1, avg_, 1);
  }
  for(i = 0; i < net->n; ++i){
    layer l = net->layers[i];
    if(l.type == YOLO || l.type == REGION || l.type == DETECTION){
      memcpy(l.output, avg_ + count, sizeof(float) * l.outputs);
      count += l.outputs;
    }
  }
  detection *dets = get_network_boxes(net, buff_[0].w, buff_[0].h, demoThresh_, demoHier_, 0, 1, nboxes, 0);
  return dets;
}

void *YoloObjectDetector::detectInThread()
{
  globalframe++;
  running_ = 1;
  float nms = .45;

  layer l = net_->layers[net_->n - 1];
  float *X = buffLetter_[(buffIndex_ + 2) % 3].data;
  network_predict(*net_, X);

//  int size_of_array = sizeof(ss)/sizeof(ss[0]);
//
//
//  for (int i=0; i < size_of_array; i++){
//      printf("%lf\n", ss[i]);
//  }
//  printf("output array size: %d\n\n", size_of_array);

  image display = buff_[(buffIndex_ + 2) % 3];
  int nboxes = 0;

  detection *dets = get_network_boxes(net_, display.w, display.h, demoThresh_, demoHier_, nullptr, 1, &nboxes, 1);

  if (nms) do_nms_sort(dets, nboxes, l.classes, nms);

  draw_detections_v3(display, dets, nboxes, demoThresh_, demoNames_, demoAlphabet_, l.classes, 0); // 1 means output classes, here I ignore

  if ( (enableConsoleOutput_)&&(globalframe%20==1) ) {
//    printf("\033[2J");
//    printf("\033[1;1H");
//    printf("\nFPS:%.1f\n",fps_);
//    printf("Objects:\n\n");
      printf("FPS:%.1f\n", fps_);
  }

  // extract the bounding boxes and send them to ROS
  int i, j;
  int count = 0;
  for (i = 0; i < nboxes; ++i) {
    float xmin = dets[i].bbox.x - dets[i].bbox.w / 2.;
    float xmax = dets[i].bbox.x + dets[i].bbox.w / 2.;
    float ymin = dets[i].bbox.y - dets[i].bbox.h / 2.;
    float ymax = dets[i].bbox.y + dets[i].bbox.h / 2.;

    if (xmin < 0)
      xmin = 0;
    if (ymin < 0)
      ymin = 0;
    if (xmax > 1)
      xmax = 1;
    if (ymax > 1)
      ymax = 1;

    // iterate through possible boxes and collect the bounding boxes
    for (j = 0; j < demoClasses_; ++j) {
      if (dets[i].prob[j]) {
        float x_center = (xmin + xmax) / 2;
        float y_center = (ymin + ymax) / 2;
        float BoundingBox_width = xmax - xmin;
        float BoundingBox_height = ymax - ymin;

        // define 2D bounding box
        // BoundingBox must be 1% size of frame (3.2x2.4 pixels)
        if (BoundingBox_width > 0.02 && BoundingBox_height > 0.02) {
          roiBoxes_[count].x = x_center;
          roiBoxes_[count].y = y_center;
          roiBoxes_[count].w = BoundingBox_width;
          roiBoxes_[count].h = BoundingBox_height;
          roiBoxes_[count].Class = j;
          roiBoxes_[count].prob = dets[i].prob[j];
          count++;
        }
      }
    }
  }

  // create array to store found bounding boxes
  // if no object detected, make sure that ROS knows that num = 0
  roiBoxes_[0].num = count;

  free_detections(dets, nboxes);
  demoIndex_ = (demoIndex_ + 1) % demoFrame_;
  running_ = 0;
  return nullptr;
}

void *YoloObjectDetector::fetchInThread()
{
  IplImage* ROS_img = getIplImage();
  ipl_into_image(ROS_img, buff_[buffIndex_]);
  {
    boost::shared_lock<boost::shared_mutex> lock(mutexImageCallback_);
    buffId_[buffIndex_] = actionId_;
  }
  if(!use_grey)
    rgbgr_image(buff_[buffIndex_]);

  letterbox_image_into(buff_[buffIndex_], net_->w, net_->h, buffLetter_[buffIndex_]);

  buff_cv_l_[(buffIndex_)] = left_rectified.clone();
  buff_cv_r_[(buffIndex_)] = right_rectified.clone();

  if(counter > 2) {

      disparityFrame[(buffIndex_ + 2) % 3] = getDepth(buff_cv_l_[(buffIndex_ + 2) % 3], buff_cv_r_[(buffIndex_ + 2) % 3]);
  }

  counter ++;
  return nullptr;
}

void *YoloObjectDetector::displayInThread()
{
  show_image_cv(buff_[(buffIndex_ + 1)%3], "YOLO V3", ipl_);
  // cv::imshow("disparity_map",disparityFrame); // * 256 / disp_size);
//  cv::imshow("left_rect", origLeft);
//  cv::imshow("right_rect", origRight);
//  int c = cvWaitKey(waitKeyDelay_);
  int c = cv::waitKey(waitKeyDelay_);
  if (c != -1) c = c%256;
  if (c == 27) {
      demoDone_ = 1;
      return nullptr;
  } else if (c == 82) {
      demoThresh_ += .02;
  } else if (c == 84) {
      demoThresh_ -= .02;
      if(demoThresh_ <= .02) demoThresh_ = .02;
  } else if (c == 83) {
      demoHier_ += .02;
  } else if (c == 81) {
      demoHier_ -= .02;
      if(demoHier_ <= .0) demoHier_ = .0;
  }
  return nullptr;
}

void YoloObjectDetector::setupNetwork(char *cfgfile, char *weightfile, char *datafile, float thresh,
                                      char **names, int classes,
                                      int delay, char *prefix, int avg_frames, float hier, int w, int h,
                                      int frames, int fullscreen)
{
  demoPrefix_ = prefix;
  demoDelay_ = delay;
  demoFrame_ = avg_frames;
  image **alphabet = load_alphabet_with_file(datafile);
  demoNames_ = names;
  demoAlphabet_ = alphabet;
  demoClasses_ = classes;
  demoThresh_ = thresh;
  demoHier_ = hier;
  fullScreen_ = fullscreen;
  printf("YOLO V3\n");
//  net_ = load_network(cfgfile, weightfile, 0);
  net_ = load_network_custom(cfgfile, weightfile, 0, 1);
//  set_batch_network(net_, 1);
  fuse_conv_batchnorm(*net_);
}

void YoloObjectDetector:: yolo()
{
  const auto wait_duration = std::chrono::milliseconds(2000);
  while (!getImageStatus()) {
    printf("Waiting for image.\n");
    if (!isNodeRunning()) {
      return;
    }
    std::this_thread::sleep_for(wait_duration);
  }

  std::thread detect_thread;
  std::thread fetch_thread;
//  std::thread depth_detect_thread;

  srand(2222222);

  int i;
  demoTotal_ = sizeNetwork(net_);
  predictions_ = (float **) calloc(demoFrame_, sizeof(float*));
  for (i = 0; i < demoFrame_; ++i){
      predictions_[i] = (float *) calloc(demoTotal_, sizeof(float));
  }
  avg_ = (float *) calloc(demoTotal_, sizeof(float));

  layer l = net_->layers[net_->n - 1];
  roiBoxes_ = (darknet_ros::RosBox_ *) calloc(l.w * l.h * l.n, sizeof(darknet_ros::RosBox_));

  IplImage* ROS_img = getIplImage();
  buff_[0] = ipl_to_image(ROS_img);
  buff_[1] = copy_image(buff_[0]);
  buff_[2] = copy_image(buff_[0]);
  buffLetter_[0] = letterbox_image(buff_[0], net_->w, net_->h);
  buffLetter_[1] = letterbox_image(buff_[0], net_->w, net_->h);
  buffLetter_[2] = letterbox_image(buff_[0], net_->w, net_->h);
  disparityFrame[0] = cv::Mat(Height_crp, Width_crp, CV_8UC1, cv::Scalar(0));
  disparityFrame[1] = cv::Mat(Height_crp, Width_crp, CV_8UC1, cv::Scalar(0));
  disparityFrame[2] = cv::Mat(Height_crp, Width_crp, CV_8UC1, cv::Scalar(0));
  buff_cv_l_[0] = camImageCopy_.clone();
  buff_cv_l_[1] = camImageCopy_.clone();
  buff_cv_l_[2] = camImageCopy_.clone();
  ipl_ = cvCreateImage(cvSize(buff_[0].w, buff_[0].h), IPL_DEPTH_8U, buff_[0].c);

  int count = 0;

  if (!demoPrefix_ && viewImage_) {
    cvNamedWindow("YOLO V3", CV_WINDOW_NORMAL);
    if (fullScreen_) {
      cvSetWindowProperty("YOLO V3", CV_WND_PROP_FULLSCREEN, CV_WINDOW_FULLSCREEN);
    } else {
      cvMoveWindow("YOLO V3", 0, 0);
      cvResizeWindow("YOLO V3", 640, 480);
    }
  }

  demoTime_ = what_time_is_it_now();

  while (!demoDone_) {
    buffIndex_ = (buffIndex_ + 1) % 3;
    fetch_thread = std::thread(&YoloObjectDetector::fetchInThread, this);
    detect_thread = std::thread(&YoloObjectDetector::detectInThread, this);

    if (!demoPrefix_) {
      fps_ = 1./(what_time_is_it_now() - demoTime_);
      demoTime_ = what_time_is_it_now();
      if (viewImage_) {
        displayInThread();
      }
      publishInThread();
    } else {
      char name[256];
      sprintf(name, "%s_%08d", demoPrefix_, count);
      save_image(buff_[(buffIndex_ + 1) % 3], name);
    }

    fetch_thread.join();
    detect_thread.join();

//    if(!disparityFrame.empty()) {
//        cv::imshow("disparity_map", disparityFrame);
//        cv::waitKey(0);
//    }
      ++count;
    if (!isNodeRunning()) {
      demoDone_ = true;
    }
  }

}

IplImage* YoloObjectDetector::getIplImage()
{
  boost::shared_lock<boost::shared_mutex> lock(mutexImageCallback_);
  auto * ROS_img = new IplImage(camImageCopy_);
  return ROS_img;
}

bool YoloObjectDetector::getImageStatus()
{
  boost::shared_lock<boost::shared_mutex> lock(mutexImageStatus_);
  return imageStatus_;
}

bool YoloObjectDetector::isNodeRunning()
{
  boost::shared_lock<boost::shared_mutex> lock(mutexNodeStatus_);
  return isNodeRunning_;
}

void *YoloObjectDetector::publishInThread()
{
  // Publish image.
  cv::Mat cvImage = cv::cvarrToMat(ipl_);
  if (!publishDetectionImage(cv::Mat(cvImage))) {
    ROS_DEBUG("Detection image has not been broadcasted.");
  }

  // Publish bounding boxes and detection result.
  int num = roiBoxes_[0].num;
  if (num > 0 && num <= 100) {
    for (int i = 0; i < num; i++) {
      for (int j = 0; j < numClasses_; j++) {
        if (roiBoxes_[i].Class == j) {
          rosBoxes_[j].push_back(roiBoxes_[i]);
          rosBoxCounter_[j]++;
        }
      }
    }

    std_msgs::Int8 msg;
    msg.data = static_cast<signed char>(num);
    objectPublisher_.publish(msg);

    for (int i = 0; i < numClasses_; i++) {
      if (rosBoxCounter_[i] > 0) {
        for (int j = 0; j < rosBoxCounter_[i]; j++) {
          auto center_c_ = static_cast<int>(rosBoxes_[i][j].x * Width_crp);     //2D column
          auto center_r_ = static_cast<int>(rosBoxes_[i][j].y * Height_crp);    //2D row

          auto xmin = (rosBoxes_[i][j].x - rosBoxes_[i][j].w / 2) * Width_crp;
          auto ymin = (rosBoxes_[i][j].y - rosBoxes_[i][j].h / 2) * Height_crp;
          auto xmax = (rosBoxes_[i][j].x + rosBoxes_[i][j].w / 2) * Width_crp;
          auto ymax = (rosBoxes_[i][j].y + rosBoxes_[i][j].h / 2) * Height_crp;

//            std::cout << "xmin: " << xmin << ", ymin: " <<ymin<<", xmax: " <<xmax<<", ymax: "<< ymax << std::endl;

          if(ymax >= Height_crp)    ymax = Height_crp - 1;
          if(xmax >= Width_crp)     xmax = Width_crp - 1;
          int median_kernel = std::min(xmax - xmin, ymax - ymin);

            // if ((xmin > 2) &&(ymin > 2) && (counter>2) ) {
          if ((counter>2) ) {

              if((classLabels_[i] == "car") || (classLabels_[i] == "bus")|| (classLabels_[i] == "motor") || (classLabels_[i] == "bike")
                 || (classLabels_[i] == "truck")  || (classLabels_[i] == "rider") || (classLabels_[i] == "person")) {

                  auto dis = static_cast<int>(Util::median_mat(disparityFrame[(buffIndex_ + 1) % 3], center_c_, center_r_, median_kernel));  // find 3x3 median
                  cv::Rect_<int> rect = cv::Rect_<int>(static_cast<int>(xmin),
                                                       static_cast<int>(ymin),
                                                       static_cast<int>(xmax - xmin),
                                                       static_cast<int>(ymax - ymin));

                  if(dis>=12) {

//                    if(dis < 12){

//                        ROS_WARN("dis too small: %d", dis);
//                    }

                      std::vector<cv::Point3f> cent_2d, cent_3d;
                      Blob outputObs(static_cast<float>(xmin),
                                     static_cast<float>(ymin),
                                     static_cast<float>(xmax - xmin),
                                     static_cast<float>(ymax - ymin));
//                    obstacle_msgs::obs outputObs;
                      outputObs.category = classLabels_[i];
                      outputObs.probability = rosBoxes_[i][j].prob;
                      outputObs.position_3d[0] = x3DPosition[center_c_][dis];
                      outputObs.position_3d[1] = y3DPosition[center_r_][dis];
                      outputObs.position_3d[2] = depth3D[dis];
                      outputObs.xmin = xmin;
                      outputObs.xmax = xmax;
                      outputObs.ymin = ymin;
                      outputObs.ymax = ymax;
//                    ROS_WARN("center 3D\nx: %f| y: %f| z: %f",
//                             outputObs.position_3d[0], outputObs.position_3d[1], depthTable[dis]);

                      double xmin_3d, xmax_3d, ymin_3d, ymax_3d;
                      xmin_3d = x3DPosition[static_cast<int>(xmin)][dis];
                      xmax_3d = x3DPosition[static_cast<int>(xmax)][dis];
                      ymin_3d = y3DPosition[static_cast<int>(ymin)][dis];
                      ymax_3d = y3DPosition[static_cast<int>(ymax)][dis];
//                    ROS_WARN("min 3D\nx: %f| y: %f", xmin_3d, xmax_3d);
//                    ROS_WARN("max 3D\nx: %f| y: %f", xmax_3d, ymax_3d);
                      outputObs.diameter = abs(static_cast<int>(xmax_3d - xmin_3d));
                      outputObs.height = abs(static_cast<int>(ymax_3d - ymin_3d));
//                    outputObs.obsHog = hog_feature;
                      outputObs.disparity = dis;
//                    obstacleBoxesResults_.obsData.push_back(outputObs);
                      currentFrameBlobs.push_back(outputObs);

                  } else {
                      std::string classname = classLabels_[i];
                      ROS_WARN("class, dis: %s, %d", classname.c_str(), dis);
                  }

              }

            } else {
//              ROS_WARN("*********************************************************");
            }

        }
      }
    }

//    std::cout<<"currentFrameBlobs: "<<currentFrameBlobs.size()<<std::endl;

//    cv::Mat beforeTracking = buff_cv_l_[(buffIndex_ + 1) % 3].clone();
//    for (auto &currentFrameBlob : currentFrameBlobs) {
//      cv::rectangle(beforeTracking, currentFrameBlob.currentBoundingRect, cv::Scalar( 0, 0, 255 ), 2);
//    }
//    cv::imshow("beforeTracking", beforeTracking);

        // TODO: wait until isDepth_new to be true
//      Tracking();
//      CreateMsg();
      roiBoxes_[0].num = 0;
//    boundingBoxesResults_.header.stamp = ros::Time::now();
//    boundingBoxesResults_.header.frame_id = "detection";
//    boundingBoxesResults_.image_header = imageHeader_;
//    boundingBoxesPublisher_.publish(boundingBoxesResults_);
  } else {
    std_msgs::Int8 msg;
    msg.data = 0;
    objectPublisher_.publish(msg);
//    std::cout << "************************************************num 0" << std::endl;
  }

//    std::cout << "************************************************new frame" << std::endl;
    cv::Mat beforeTracking = buff_cv_l_[(buffIndex_ + 1) % 3].clone();
//    cv::imshow("beforeTracking", beforeTracking);

  ObstacleDetector.ExecuteDetection(disparityFrame[(buffIndex_ + 1) % 3], beforeTracking);
    Tracking();
    CreateMsg();

  obstacleBoxesResults_.header.stamp = image_time_;
  obstacleBoxesResults_.header.frame_id = pub_obs_frame_id;
  obstacleBoxesResults_.real_header.stamp = ros::Time::now();
  obstacleBoxesResults_.real_header.frame_id = pub_obs_frame_id;
  obstaclePublisher_.publish(obstacleBoxesResults_);

  obstacleBoxesResults_.obsData.clear();
  for (int i = 0; i < numClasses_; i++) {
    rosBoxes_[i].clear();
    rosBoxCounter_[i] = 0;
  }

  return nullptr;
}

void YoloObjectDetector::matchCurrentFrameBlobsToExistingBlobs() {

    int tracksOrMatHeight = (int)blobs.size();
    int detsOrMatWidth = (int)currentFrameBlobs.size();
//    cv::Mat appDisSimilarity(tracksOrMatHeight, detsOrMatWidth, CV_64FC1, cv::Scalar(1.0));
//    cv::Mat motionDisSimilarity(tracksOrMatHeight, detsOrMatWidth, CV_64FC1, cv::Scalar(1.0));
    cv::Mat disSimilarity(tracksOrMatHeight, detsOrMatWidth, CV_64FC1, cv::Scalar(1.0));

    for (int c=0; c<detsOrMatWidth; c++) {

        Blob currBlob = currentFrameBlobs[c];

        for (int r = 0; r < tracksOrMatHeight; r++) {
            Blob blob = blobs[r];
            if (blob.blnStillBeingTracked) {
                if (currBlob.category == blob.category) {

                    cv::Rect predRect;
                    predRect.width = static_cast<int>(blob.state.at<float>(4));
                    predRect.height = static_cast<int>(blob.state.at<float>(5));
                    predRect.x = static_cast<int>(blob.state.at<float>(0) - predRect.width / 2);
                    predRect.y = static_cast<int>(blob.state.at<float>(1) - predRect.height / 2);

//                    std::cout<< r <<" predRect: "<< predRect <<", "<<c<<" detectRect: " << currBlob.currentBoundingRect <<std::endl;

                    cv::Rect intersection = predRect & currBlob.currentBoundingRect;//currBlob.boundingRects.back();
                    cv::Rect unio = predRect | currBlob.currentBoundingRect;//currBlob.boundingRects.back();
                    disSimilarity.at<double>(r,c) = 1.0 - (double)intersection.area()/unio.area();

                }
            }
        }
    }

//    double min, max;
//    cv::minMaxLoc(disSimilarity, &min, &max);
//        double thForHungarianCost = std::max(0.75,max*0.5);

    std::vector< std::vector<double> > costMatrix;
    for (int r = 0; r < tracksOrMatHeight; r++)  {
        std::vector<double> costForEachTrack;
        for (int c=0; c<detsOrMatWidth; c++) {
            costForEachTrack.push_back(disSimilarity.at<double>(r,c));
        }
        costMatrix.push_back(costForEachTrack);
    }

//        std::cout<<"costMatrix: "<<costMatrix.size()<<", "<<costMatrix[0].size()<<"; simHeight: "<<simHeight<<", simWidth: "<<simWidth<<std::endl;

    HungarianAlgorithm HungAlgo;
    std::vector<int> assignment;

    double hungarianCost = HungAlgo.Solve(costMatrix, assignment);
//        std::cout<<"hungarianCost: "<<hungarianCost<<std::endl;

    for (int trackID = 0; trackID < costMatrix.size(); trackID++){
//            std::cout << trackID << "," << assignment[trackID] << "\t";
        if (assignment[trackID]>-1) {
            Blob &currentFrameBlob = currentFrameBlobs.at(static_cast<unsigned long>(assignment[trackID]));
            double disSimValue = disSimilarity.at<double>(trackID,assignment[trackID]);
            if ( (!blobs[trackID].blnAlreadyTrackedInThisFrame) && disSimValue<0.7 ) { //(minDisSimilarity < max)
                currentFrameBlob.blnAlreadyTrackedInThisFrame = true;
                addBlobToExistingBlobs(currentFrameBlob, blobs, trackID);
            } else {
                addNewBlob(currentFrameBlob, blobs);
            }
        }
    }
//        std::cout<<std::endl;

    for (int c=0; c<detsOrMatWidth; c++){
        Blob &currentFrameBlob = currentFrameBlobs.at(c);
        if(!currentFrameBlob.blnAlreadyTrackedInThisFrame)
            addNewBlob(currentFrameBlob, blobs);
    }

    for (auto &existingBlob : blobs) {
        if (!existingBlob.blnCurrentMatchFoundOrNewBlob) {
            existingBlob.intNumOfConsecutiveFramesWithoutAMatch++;
        }
        if (existingBlob.intNumOfConsecutiveFramesWithoutAMatch >= 50) {
            existingBlob.blnStillBeingTracked = false;
        }
    }

// std::cout<<"Debug matchCurrentFrameBlobsToExistingBlobs 5"<<std::endl;

}

void YoloObjectDetector::addBlobToExistingBlobs(Blob &currentFrameBlob, std::vector<Blob> &existingBlobs, int &intIndex) {

    existingBlobs[intIndex].currentBoundingRect = currentFrameBlob.currentBoundingRect;
    existingBlobs[intIndex].centerPositions.push_back(currentFrameBlob.centerPositions.back());
    existingBlobs[intIndex].dblCurrentDiagonalSize = currentFrameBlob.dblCurrentDiagonalSize;
    existingBlobs[intIndex].probability = currentFrameBlob.probability;
    existingBlobs[intIndex].disparity = currentFrameBlob.disparity;
    existingBlobs[intIndex].position_3d = currentFrameBlob.position_3d;

    existingBlobs[intIndex].xmin = currentFrameBlob.xmin;
    existingBlobs[intIndex].xmax = currentFrameBlob.xmax;
    existingBlobs[intIndex].ymin = currentFrameBlob.ymin;
    existingBlobs[intIndex].ymax = currentFrameBlob.ymax;

    existingBlobs[intIndex].blnStillBeingTracked = true;
    existingBlobs[intIndex].blnCurrentMatchFoundOrNewBlob = true;
    existingBlobs[intIndex].blnAlreadyTrackedInThisFrame = true;
    existingBlobs[intIndex].counter = currentFrameBlob.counter + 1;
    existingBlobs[intIndex].intNumOfConsecutiveFramesWithoutAMatch =0;

    //update motion model
    existingBlobs[intIndex].meas.at<float>(0) = currentFrameBlob.meas.at<float>(0);
    existingBlobs[intIndex].meas.at<float>(1) = currentFrameBlob.meas.at<float>(1);
    existingBlobs[intIndex].meas.at<float>(2) = currentFrameBlob.meas.at<float>(2);
    existingBlobs[intIndex].meas.at<float>(3) = currentFrameBlob.meas.at<float>(3);
    existingBlobs[intIndex].kf.correct(existingBlobs[intIndex].meas); // Kalman Correction

}

void YoloObjectDetector::addNewBlob(Blob &currentFrameBlob, std::vector<Blob> &existingBlobs) {

    currentFrameBlob.blnCurrentMatchFoundOrNewBlob = true;

    currentFrameBlob.blnStillBeingTracked = true;

    existingBlobs.push_back(currentFrameBlob);
}

void YoloObjectDetector::Tracking (){

    if (blnFirstFrame) {
        if (!currentFrameBlobs.empty()){
            blnFirstFrame = false;
            for (auto &currentFrameBlob : currentFrameBlobs)
                blobs.push_back(currentFrameBlob);
        }
    } else {
        for (auto &existingBlob : blobs) {
            existingBlob.blnCurrentMatchFoundOrNewBlob = false;
            existingBlob.blnAlreadyTrackedInThisFrame = false;
            // >>>> Matrix A
            auto dT = static_cast<float>(0.04 + (0.04 * existingBlob.intNumOfConsecutiveFramesWithoutAMatch));
            existingBlob.kf.transitionMatrix.at<float>(2) = dT;//dT;
            existingBlob.kf.transitionMatrix.at<float>(9) = dT;//dT;
            // <<<< Matrix A
            existingBlob.state = existingBlob.kf.predict();
        }

//            std::cout<<"blob prediction finished"<<std::endl;

        if (!currentFrameBlobs.empty()){
            matchCurrentFrameBlobsToExistingBlobs();
        } else {
            for (auto &existingBlob : blobs) {
                if (!existingBlob.blnCurrentMatchFoundOrNewBlob) {
                    existingBlob.intNumOfConsecutiveFramesWithoutAMatch++;
                }
                if (existingBlob.intNumOfConsecutiveFramesWithoutAMatch >= 50) {
                    existingBlob.blnStillBeingTracked = false;
                    //blobs.erase(blobs.begin() + i);
                }
            }
        }
//            std::cout<<"blob association finished"<<std::endl;
    }

//    if (blnFirstFrame) {
//        blnFirstFrame = false;
//        for (auto &currentFrameBlob : currentFrameBlobs)
//            blobs.push_back(currentFrameBlob);
//    } else
//        matchCurrentFrameBlobsToExistingBlobs();

    currentFrameBlobs.clear();
}

void YoloObjectDetector::CreateMsg(){
    cv::Mat color_out;

    cv::Mat output1 = disparityFrame[(buffIndex_ + 1) % 3].clone();
    cv::Mat output = buff_cv_l_[(buffIndex_ + 1) % 3].clone();
    if(output.type() == CV_8UC1)
        cv::cvtColor(output, color_out, CV_GRAY2RGB);
    else
        color_out = output;

    std::vector<cv::Scalar> colors;
    cv::RNG rng(0);
    for(int i=0; i < blobs.size(); i++)
        colors.push_back(cv::Scalar(rng.uniform(0,255), rng.uniform(0, 255), rng.uniform(0, 255)));

    for (long int i = 0; i < blobs.size(); i++) {
//            if (blobs[i].blnStillBeingTracked == true) {
        if (blobs[i].blnCurrentMatchFoundOrNewBlob) {
            cv::rectangle(color_out, blobs[i].currentBoundingRect, colors.at(i), 2);
            int rectMinX = blobs[i].currentBoundingRect.x;
            int rectMinY = blobs[i].currentBoundingRect.y;
            cv::rectangle(color_out, cv::Rect(rectMinX, rectMinY, 40, 20), colors.at(i), CV_FILLED);
            std::ostringstream str;
            // str << blobs[i].position_3d[2] <<"m, ID="<<i<<"; "<<blobs[i].disparity;
            str << i;
            cv::putText(color_out, str.str(), cv::Point(rectMinX, rectMinY+16) , CV_FONT_HERSHEY_PLAIN, 1.5, CV_RGB(255,255,255));

            cv::rectangle(output1, blobs[i].currentBoundingRect, cv::Scalar( 255, 255, 255 ), 2);
        }
    }
    if(viewImage_) {
      cv::imshow("debug", color_out);
      cv::imshow("disparity", output1);
      // cv::waitKey(0);
    }
    frame_num ++;
    if(enableEvaluation_){
    sprintf(s, "f%03d.txt", frame_num);
    sprintf(im, "f%03d.png", frame_num);
    file_name = ros::package::getPath("cubicle_detect") + "/seq_1/results/" + s;
    img_name = ros::package::getPath("cubicle_detect") + "/seq_1/" + im;

    file.open(file_name.c_str(), std::ios::app);
    }

    int cate = 0;

    for (unsigned long int i = 0; i < blobs.size(); i++) {

        if (blobs[i].blnCurrentMatchFoundOrNewBlob) {

            if((blobs[i].category == "car") || (blobs[i].category == "bus")|| (blobs[i].category == "motor")
                || (blobs[i].category == "truck")  || (blobs[i].category == "rider") || (blobs[i].category == "person")
                || (blobs[i].category == "train") ) {

                if(enableEvaluation_){
                    if((blobs[i].category == "car") || (blobs[i].category == "bus")|| (blobs[i].category == "motor")
                       || (blobs[i].category == "truck")  || (blobs[i].category == "rider")
                       || (blobs[i].category == "train") )
                        cate = 0;
                    else if(blobs[i].category == "person")
                        cate = 1;
                }
                obstacle_msgs::obs tmpObs;

                tmpObs.identityID = i;

                tmpObs.centerPos.x = blobs[i].position_3d[0];
                tmpObs.centerPos.y = blobs[i].position_3d[1];
                tmpObs.centerPos.z = blobs[i].position_3d[2];
                tmpObs.diameter = blobs[i].diameter;
                tmpObs.height = blobs[i].height;
                tmpObs.xmin = blobs[i].xmin;
                tmpObs.ymin = blobs[i].ymin;
                tmpObs.xmax = blobs[i].xmax;
                tmpObs.ymax = blobs[i].ymax;

                tmpObs.counter = blobs[i].counter;
                tmpObs.classes = blobs[i].category;
                tmpObs.probability = blobs[i].probability;
//            tmpObs.histogram = blobs[i].obsHog;

                obstacleBoxesResults_.obsData.push_back(tmpObs);


//            ROS_WARN("center ID: %d | type: %s\nx: %f| y: %f| z: %f \n", i, tmpObs.classes,
//                    tmpObs.centerPos.x, tmpObs.centerPos.y, tmpObs.centerPos.z);


                ////*--------------Generate Evaluation files----------------------*////
                if(enableEvaluation_){
                    file << i << " " << blobs[i].currentBoundingRect.x << " " << blobs[i].currentBoundingRect.y << " "
                         << blobs[i].currentBoundingRect.x + blobs[i].currentBoundingRect.width << " " <<
                         blobs[i].currentBoundingRect.y + blobs[i].currentBoundingRect.height << " " << cate
                         << std::endl;
                }
            }
        }
    }
    if(enableEvaluation_){
        file.close();
        cv::imwrite(img_name, buff_cv_l_[(buffIndex_ + 1) % 3]);
    }
}


} /* namespace darknet_ros*/
